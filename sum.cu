#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "include/cuda_stl.h"

// cpu
void sumArrays(float * a,float * b,float * res,const int size) {
    for(int i=0;i<size;i++) {
        res[i]=a[i]+b[i];
        // printf("sumArrays arrary index is: %d , sum is: %f\n", i, res[i]);//查看耗时，注释掉
    }
}

//通常grid和block为一维，apollo中pointpillar也是这样
// 1 2 3 4 5    1 2 3 4 5    1 2 3 4 5    ...
__global__ void sumArrayGpu(float *a, float *b, float *res, int size) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    res[i] = a[i] + b[i];
    // printf("arrary index is: %d , sum is: %f\n", i, res[i]);//查看耗时，注释掉
}


int main(int arg, char **argv) {
    int dev = 0;
    hipSetDevice(dev);

    int nBytes = sizeof(float) * SIZE;

// cpu mem
    float *a_h = (float *)malloc(nBytes);
    float *b_h = (float *)malloc(nBytes);
    float *res_h = (float *)malloc(nBytes);
    memset(res_h,0,nBytes);

// GPU mem
    float *a_d, *b_d, *res_d;
    hipMalloc((float **)&a_d, nBytes);
    hipMalloc((float **)&b_d, nBytes);
    hipMalloc((float **)&res_d, nBytes);

// init data
    initData(a_h, SIZE);
    initData(b_h, SIZE);
    
    hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice);

    // sumArrays(a_h, b_h, res_h, SIZE);

    dim3 block(1024);
    dim3 grid(SIZE/block.x);
    sumArrayGpu<<<grid, block>>>(a_d, b_d, res_d, SIZE);

    hipMemcpy(res_h, res_d, nBytes, hipMemcpyDeviceToHost);

    hipDeviceReset();

    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);
    free(a_h);
    free(b_h);
    free(res_h);

    return 0;
}