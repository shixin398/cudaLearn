/*
*1_check_dimension
*/
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void checkIndex(void)
{
    printf("x index: %d, threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d) gridDim(%d,%d,%d)\n",
    threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y + blockIdx.y*blockDim.x*blockDim.y*gridDim.x,
    threadIdx.x,threadIdx.y,threadIdx.z,
    blockIdx.x,blockIdx.y,blockIdx.z,
    blockDim.x,blockDim.y,blockDim.z,
    gridDim.x,gridDim.y,gridDim.z);

    printf("------------.\n");
}
int main(int argc,char **argv)
{
    dim3 block(3,3);
    dim3 grid(3,3);
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x,grid.y,grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x,block.y,block.z);
    checkIndex<<<grid,block>>>();
    hipDeviceReset();
    return 0;
}